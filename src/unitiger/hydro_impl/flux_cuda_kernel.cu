
#include <hip/hip_runtime.h>
#ifdef OCTOTIGER_HAVE_CUDA

#include <buffer_manager.hpp>
#include <cuda_buffer_util.hpp>
#include "octotiger/options.hpp"
#include "octotiger/cuda_util/cuda_helper.hpp"
#include <cuda_runtime.h>
#include <stream_manager.hpp>

#include "octotiger/unitiger/hydro_impl/flux_kernel_interface.hpp"

#include <mutex>

__device__ inline int flip_dim(const int d, const int flip_dim) {
		int dims[3];
		int k = d;
		for (int dim = 0; dim < 3; dim++) {
			dims[dim] = k % 3;
			k /= 3;
		}
		k = 0;
		dims[flip_dim] = 2 - dims[flip_dim];
		for (int dim = 0; dim < 3; dim++) {
			k *= 3;
			k += dims[2 - dim];
		}
		return k;
}

__device__ const int faces[3][9] = { { 12, 0, 3, 6, 9, 15, 18, 21, 24 }, { 10, 0, 1, 2, 9, 11,
			18, 19, 20 }, { 4, 0, 1, 2, 3, 5, 6, 7, 8 } };

__device__ const int xloc[27][3] = {
	/**/{ -1, -1, -1 }, { +0, -1, -1 }, { +1, -1, -1 },
	/**/{ -1, +0, -1 }, { +0, +0, -1 }, { 1, +0, -1 },
	/**/{ -1, +1, -1 }, { +0, +1, -1 }, { +1, +1, -1 },
	/**/{ -1, -1, +0 }, { +0, -1, +0 }, { +1, -1, +0 },
	/**/{ -1, +0, +0 }, { +0, +0, +0 }, { +1, +0, +0 },
	/**/{ -1, +1, +0 }, { +0, +1, +0 }, { +1, +1, +0 },
	/**/{ -1, -1, +1 }, { +0, -1, +1 }, { +1, -1, +1 },
	/**/{ -1, +0, +1 }, { +0, +0, +1 }, { +1, +0, +1 },
	/**/{ -1, +1, +1 }, { +0, +1, +1 }, { +1, +1, +1 } };

__device__ const double quad_weights[9] = { 16. / 36., 1. / 36., 4. / 36., 1. / 36., 4. / 36., 4.
			/ 36., 1. / 36., 4. / 36., 1. / 36. };

std::once_flag flag1;

__host__ void init_gpu_masks(bool *masks) {
  auto masks_boost = create_masks();
  cudaMemcpy(masks, masks_boost.data(), NDIM * 1000 * sizeof(bool), cudaMemcpyHostToDevice);
}

__host__ const bool* get_gpu_masks(void) {
    static recycler::cuda_device_buffer<bool> masks(NDIM * 1000, 0);
    std::call_once(flag1, init_gpu_masks, masks.device_side_buffer);
    return masks.device_side_buffer;
}

__device__ const int offset = 0;
__device__ const int compressedH_DN[3] = {100, 10, 1};
__device__ const int face_offset = 27 * 1000;
__device__ const int dim_offset = 1000;

__global__ void
__launch_bounds__(900, 1)
 flux_cuda_kernel(const double * __restrict__ q_combined, const double * __restrict__ x_combined, double * __restrict__ f_combined,
    double * amax, int * amax_indices, int * amax_d, const bool * __restrict__ masks, const double omega, const double dx, const double A_, const double B_, const double fgamma, const double de_switch_1) {
  __shared__ double sm_amax[900];
  __shared__ int sm_d[900];
  __shared__ int sm_i[900];

  // 3 dim 1000 i workitems
  const int dim = blockIdx.z;
  const int index = threadIdx.x * 100 + threadIdx.y * 10 + threadIdx.z + 100;
  int tid = index - 100;   
  //if(tid == 0)
  // printf("starting...");
  const int nf = 15;

  double local_f[15] = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
  double local_x[3] = {0.0, 0.0, 0.0};
  double local_vg[3] = {0.0, 0.0, 0.0};
  for (int f = 0; f < nf; f++) {
      f_combined[dim * 15 * 1000 + f * 1000 + index] = 0.0;
  }

  double mask = masks[index + dim * dim_offset];
  double current_amax = 0.0;
  int current_d = 0;
  for (int fi = 0; fi < 9; fi++) {    // 9
    double this_ap = 0.0, this_am = 0.0;    // tmps
    const int d = faces[dim][fi];
    const int flipped_dim = flip_dim(d, dim);
    for (int dim = 0; dim < 3; dim++) {
        local_x[dim] = x_combined[dim * 1000 + index] + (0.5 * xloc[d][dim] * dx);
    }
    local_vg[0] = -omega * (x_combined[1000 + index] + 0.5 * xloc[d][1] * dx);
    local_vg[1] = +omega * (x_combined[index] + 0.5 * xloc[d][0] * dx);
    local_vg[2] = 0.0;
    /*if (index == 111 && dim == 0) {
      printf("CUDAInput: Q1i %i Q2i %i :: X2 %f X1 %f X0 %f :: vg2 %f vg1 %f vg0 %f dx: %f\n",dim_offset * d + index,  dim_offset * flipped_dim - compressedH_DN[dim] + index, local_x[2], local_x[1], local_x[0], local_vg[2], local_vg[1], local_vg[0] ,dx);
    }*/
    inner_flux_loop2<double>(omega, nf, A_, B_, q_combined, local_f, local_x, local_vg,
      this_ap, this_am, dim, d, dx, fgamma, de_switch_1, dim_offset * d + index, dim_offset * flipped_dim - compressedH_DN[dim] + index, face_offset);
    this_ap *= mask;
    this_am *= mask;
    const double amax_tmp = max_wrapper(this_ap, (-this_am));
    if (amax_tmp > current_amax) {
      current_amax = amax_tmp;
      current_d = d;
    }
    for (int f = 0; f < nf; f++) {
      f_combined[dim * 15 * 1000 + f * 1000 + index] += quad_weights[fi] * local_f[f] * mask;
    }
 }

 // Find maximum:
 sm_amax[tid] = current_amax;
 sm_d[tid] = current_d;
 sm_i[tid] = tid;
 __syncthreads();
 // First step as we do not have multiples of 2
 if(tid < 450) {
   if (sm_amax[tid + 450 ] > sm_amax[tid]) {
     sm_amax[tid] = sm_amax[tid + 450];
     sm_d[tid] = sm_d[tid + 450];
     sm_i[tid] = sm_i[tid + 450];
   }
 }
 __syncthreads();
 // Max reduction with multiple warps
 for (int tid_border = 256; tid_border >= 32; tid_border /= 2) {
   if(tid < tid_border) {
     if (sm_amax[tid + tid_border] > sm_amax[tid]) {
       sm_amax[tid] = sm_amax[tid + tid_border];
       sm_d[tid] = sm_d[tid + tid_border];
       sm_i[tid] = sm_i[tid + tid_border];
     }
   }
   __syncthreads();
 }
 // Max reduction within one warps
 for (int tid_border = 16; tid_border >= 1; tid_border /= 2) {
   if(tid < tid_border) {
     if (sm_amax[tid + tid_border] > sm_amax[tid]) {
       sm_amax[tid] = sm_amax[tid + tid_border];
       sm_d[tid] = sm_d[tid + tid_border];
       sm_i[tid] = sm_i[tid + tid_border];
     }
   }
 }

 if (tid == 0) {
   amax[dim] = sm_amax[0];
   amax_indices[dim] = sm_i[0];
   amax_d[dim] = sm_d[0];
 //printf("%i dim: %f %i %i \n", dim, amax[dim], amax_indices[dim], amax_d[dim]);
 }


 return;
}

timestep_t launch_flux_cuda(const hydro::recon_type<NDIM>& Q, hydro::flux_type& F, hydro::x_type& X,
    safe_real omega, const size_t nf_) {
    timestep_t ts;

    // Check availability
    bool avail = stream_pool::interface_available<hpx::cuda::experimental::cuda_executor,
                 pool_strategy>(opts().cuda_buffer_capacity);
  
    // Call CPU kernel as no stream is free
    if (!avail) {
       return flux_cpu_kernel(Q, F, X, omega, nf_);
    } else {

    size_t device_id =
      stream_pool::get_next_device_id<hpx::cuda::experimental::cuda_executor,
      pool_strategy>();

    stream_interface<hpx::cuda::experimental::cuda_executor, pool_strategy> executor;

    std::vector<double, recycler::recycle_allocator_cuda_host<double>> combined_q(
        15 * 27 * 10 * 10 * 10 + 32);
    auto it = combined_q.begin();
    for (auto face = 0; face < 15; face++) {
        for (auto d = 0; d < 27; d++) {
            auto start_offset = 2 * 14 * 14 + 2 * 14 + 2;
            for (auto ix = 2; ix < 2 + INX + 2; ix++) {
                for (auto iy = 2; iy < 2 + INX + 2; iy++) {
                    it = std::copy(Q[face][d].begin() + start_offset,
                        Q[face][d].begin() + start_offset + 10, it);
                    start_offset += 14;
                }
                start_offset += (2 + 2) * 14;
            }
        }
    }
    recycler::cuda_device_buffer<double> device_q(15 * 27 * 10 * 10 * 10 + 32, device_id);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
    cudaMemcpyAsync, device_q.device_side_buffer,
    combined_q.data(), (15 * 27 * 10 * 10 * 10 + 32) * sizeof(double), cudaMemcpyHostToDevice);

    std::vector<double, recycler::recycle_allocator_cuda_host<double>> combined_x(NDIM * 1000 + 32);
    auto it_x = combined_x.begin();
    for (size_t dim = 0; dim < NDIM; dim++) {
      auto start_offset = 2 * 14 * 14 + 2 * 14 + 2;
      for (auto ix = 2; ix < 2 + INX + 2; ix++) {
          for (auto iy = 2; iy < 2 + INX + 2; iy++) {
              it_x = std::copy(X[dim].begin() + start_offset,
                  X[dim].begin() + start_offset + 10, it_x);
              start_offset += 14;
          }
          start_offset += (2 + 2) * 14;
      }
    }
    const cell_geometry<3, 8> geo;
    double dx = X[0][geo.H_DNX] - X[0][0];
    recycler::cuda_device_buffer<double> device_x(NDIM * 1000 + 32, device_id);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
    cudaMemcpyAsync, device_x.device_side_buffer,
    combined_x.data(), (NDIM * 1000 + 32) * sizeof(double), cudaMemcpyHostToDevice);

    std::vector<double, recycler::recycle_allocator_cuda_host<double>> combined_f(NDIM * 15 * 1000 + 32);
    recycler::cuda_device_buffer<double> device_f(NDIM * 15 * 1000 + 32, device_id);
    const bool *masks = get_gpu_masks();

    recycler::cuda_device_buffer<double> device_amax(NDIM);
    recycler::cuda_device_buffer<int> device_amax_indices(NDIM);
    recycler::cuda_device_buffer<int> device_amax_d(NDIM);
    double A_ = physics<NDIM>::A_;
    double B_ = physics<NDIM>::B_;
    double fgamma = physics<NDIM>::fgamma_;
    double de_switch_1 = physics<NDIM>::de_switch_1;

    dim3 const grid_spec(1, 1, 3);
    dim3 const threads_per_block(9, 10, 10);
    void* args[] = {&(device_q.device_side_buffer),
      &(device_x.device_side_buffer), &(device_f.device_side_buffer), &(device_amax.device_side_buffer),
      &(device_amax_indices.device_side_buffer), &(device_amax_d.device_side_buffer), &masks, &omega, &dx, &A_, &B_, &fgamma, &de_switch_1};
    executor.post(
    cudaLaunchKernel<decltype(flux_cuda_kernel)>,
    flux_cuda_kernel, grid_spec, threads_per_block, args, 0);

    // Move data to host
    std::vector<double, recycler::recycle_allocator_cuda_host<double>> amax(NDIM);
    std::vector<int, recycler::recycle_allocator_cuda_host<int>> amax_indices(NDIM);
    std::vector<int, recycler::recycle_allocator_cuda_host<int>> amax_d(NDIM);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, amax.data(),
               device_amax.device_side_buffer, NDIM * sizeof(double),
               cudaMemcpyDeviceToHost);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, amax_indices.data(),
               device_amax_indices.device_side_buffer, NDIM * sizeof(int),
               cudaMemcpyDeviceToHost);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, amax_d.data(),
               device_amax_d.device_side_buffer, NDIM * sizeof(int),
               cudaMemcpyDeviceToHost);
    auto fut = hpx::async(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, combined_f.data(), device_f.device_side_buffer,
               (NDIM * 15 * 1000 + 32) * sizeof(double), cudaMemcpyDeviceToHost);
    fut.get();
    /*std::cout << "cuda kernel:" << std::endl;
    for (size_t dim = 0; dim < 1; dim++) {
        for (auto face = 0; face < 1; face++) {
          for (auto i = 111; i < 120; i++) {
            std::cout << combined_f[i] << " ";
          }
        }
        std::cout << std::endl << std::endl;
    } 
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaStreamSynchronize);
    std::cout << "ended cuda kernel:" << std::endl;
    std::cin.get();*/
    // Convert data back to Octo-Tiger format
    for (size_t dim = 0; dim < NDIM; dim++) {
        for (auto face = 0; face < 15; face++) {
            auto face_offset = dim * 15 * 1000 + face * 1000;
            auto start_offset = 2 * 14 * 14 + 2 * 14 + 2;
            auto compressed_offset = 0;
            for (auto ix = 2; ix < 2 + INX + 2; ix++) {
                for (auto iy = 2; iy < 2 + INX + 2; iy++) {
                    std::copy(combined_f.begin() + face_offset + compressed_offset,
                        combined_f.begin() + face_offset + compressed_offset + 10,
                        F[dim][face].data() + start_offset);
                    compressed_offset += 10;
                    start_offset += 14;
                }
                start_offset += (2 + 2) * 14;
            }
        }
    }
    // Find Maximum
    size_t current_dim = 0;
    for (size_t dim_i = 1; dim_i < NDIM; dim_i++) {
      if (amax[dim_i] > amax[current_dim]) { 
        current_dim = dim_i;
      }
    }
    //std::cin.get();
    static thread_local std::vector<double> URs(nf_), ULs(nf_);
    const size_t current_max_index = amax_indices[current_dim];
    const size_t current_d = amax_d[current_dim];
    ts.a = amax[current_dim];
    ts.x = combined_x[current_max_index];
    ts.y = combined_x[current_max_index + 1000];
    ts.z = combined_x[current_max_index + 2000];
    const auto flipped_dim = geo.flip_dim(current_d, current_dim);
    constexpr int compressedH_DN[3] = {100, 10, 1};
    for (int f = 0; f < nf_; f++) {
        URs[f] = combined_q[current_max_index + f * face_offset + dim_offset * current_d];
        ULs[f] = combined_q[current_max_index - compressedH_DN[current_dim] + f * face_offset +
            dim_offset * flipped_dim];
    }
    ts.ul = ULs;
    ts.ur = URs;
    ts.dim = current_dim;
    return ts;
    }
}


#endif